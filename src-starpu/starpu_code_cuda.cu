#include "hip/hip_runtime.h"
#include <starpu.h>
#include "mytype.h"

#define MAXATOMS 64
#define MAXTHREADS 64

static __global__ void do_ljForce(
    real_t s6, real_t eShift, real_t epsilon, real_t rCut2, int nNbrBoxes,
    int nLocalBoxes, int* nbrBoxes, int* nAtoms, real3* r, real3* f, real_t* U,
    real_t* ePot, size_t nbrBoxes_offset, size_t nbrBoxes_nx,
    size_t iOff_offset
){
    extern __shared__ real_t ePot_data[];
    unsigned int tid = threadIdx.x;
    
    ePot_data[tid] = 0.0;

    //*ePot = 0.0;
    int iBox = nbrBoxes_offset + tid;
    //for (int iBox = nbrBoxes_offset; iBox < nbrBoxes_offset + nbrBoxes_nx && iBox < nLocalBoxes; iBox++){

      int nIBox = nAtoms[iBox];
      
      // loop over neighbors of iBox
      for (int jTmp=0; jTmp<nNbrBoxes; jTmp++){
            int jBox = nbrBoxes[(iBox - nbrBoxes_offset) * nNbrBoxes + jTmp];
            
            assert(jBox>=0);
            
            int nJBox = nAtoms[jBox];
            
            // loop over atoms in iBox
            for (int iOff=MAXATOMS*iBox; iOff<(iBox*MAXATOMS+nIBox); iOff++){
               // iOff traduzido para uso em dados particionados
               int task_iOff = iOff - iOff_offset;

               // loop over atoms in jBox
               for (int jOff=jBox*MAXATOMS; jOff<(jBox*MAXATOMS+nJBox); jOff++){
                  real3 dr;
                  real_t r2 = 0.0;
                  for (int m=0; m<3; m++){
                     dr[m] = r[iOff][m]-r[jOff][m];
                     r2+=dr[m]*dr[m];
                  }

                  if ( r2 <= rCut2 && r2 > 0.0){

                     // Important note:
                     // from this point on r actually refers to 1.0/r
                     r2 = 1.0/r2;
                     real_t r6 = s6 * (r2*r2*r2);
                     real_t eLocal = r6 * (r6 - 1.0) - eShift;
                     U[task_iOff] += 0.5*eLocal;
                     //*ePot = *ePot + 0.5*eLocal;
                     ePot_data[tid] += 0.5*eLocal;

                     // different formulation to avoid sqrt computation
                     real_t fr = - 4.0*epsilon*r6*r2*(12.0*r6 - 6.0);
                     for (int m=0; m<3; m++)
                           f[task_iOff][m] -= dr[m]*fr;
                  }
               } // loop over atoms in jBox
         } // loop over atoms in iBox
      } // loop over neighbor boxes
   // }
    __syncthreads();
    // do reduction in shared mem
    for (unsigned int s=1; s < blockDim.x; s *= 2)
        if ((tid % (2*s) == 0) && (tid + s < blockDim.x))
            ePot_data[tid] += ePot_data[tid + s];
        __syncthreads();
    if (tid == 0)
        *ePot += ePot_data[0];
    return;
}

extern "C" void cuda_func(void *buffers[], void *cl_arg){
    // Angariando parâmetros
    real_t s6, eShift, epsilon, rCut2;
    int    nNbrBoxes, nLocalBoxes, id;
    
    starpu_codelet_unpack_args(cl_arg, &s6, &eShift, &epsilon, &rCut2, &nNbrBoxes, &nLocalBoxes, &id);
    
    // Angariando buffers
    int* nbrBoxes = (   int*) STARPU_VECTOR_GET_PTR(buffers[0]);
    int*   nAtoms = (   int*) STARPU_VECTOR_GET_PTR(buffers[1]);
    real3*      r = ( real3*) STARPU_VECTOR_GET_PTR(buffers[2]);
    real3*      f = ( real3*) STARPU_VECTOR_GET_PTR(buffers[3]);
    real_t*     U = (real_t*) STARPU_VECTOR_GET_PTR(buffers[4]);
    real_t*  ePot = (real_t*) STARPU_VARIABLE_GET_PTR(buffers[5]);
    
    // Angariando offsets e números de elementos
    //size_t nbrBoxes_offset = (size_t) STARPU_VECTOR_GET_OFFSET(buffers[0]);
    size_t nbrBoxes_nx = (size_t) STARPU_VECTOR_GET_NX(buffers[0]);
    //size_t iOff_offset = (size_t) STARPU_VECTOR_GET_OFFSET(buffers[4]);
    size_t f_nx = (size_t) STARPU_VECTOR_GET_NX(buffers[3]);
    size_t U_nx = (size_t) STARPU_VECTOR_GET_NX(buffers[4]);

    // Conferindo se offsets e tamanhos estão dentro do esperado
    //STARPU_ASSERT((nbrBoxes_offset / sizeof(int)) % nNbrBoxes == 0);
    STARPU_ASSERT(nbrBoxes_nx % nNbrBoxes == 0);
    //STARPU_ASSERT((iOff_offset / sizeof(real_t)) % MAXATOMS == 0);
    STARPU_ASSERT(U_nx % MAXATOMS == 0);
    STARPU_ASSERT(f_nx % MAXATOMS == 0);
    
    // Calculando offsets e tamanhos reais
    //nbrBoxes_offset /= nNbrBoxes * sizeof(int);
    size_t nbrBoxes_offset = (nbrBoxes_nx/nNbrBoxes) * id;
    nbrBoxes_nx     /= nNbrBoxes;
    size_t iOff_offset = U_nx * id;
    //iOff_offset     /= sizeof(real_t);

    // int n_threads = STARPU_MIN(MAXTHREADS, nbrBoxes_nx);
    // int loops_per_thread = (nbrBoxes_nx + n_threads - 1) / n_threads;
    
	do_ljForce<<<1, nbrBoxes_nx, nbrBoxes_nx * sizeof(real_t), starpu_cuda_get_local_stream()>>>(s6, eShift, epsilon, rCut2, nNbrBoxes, nLocalBoxes, nbrBoxes, nAtoms, r, f, U, ePot, nbrBoxes_offset, nbrBoxes_nx, iOff_offset);
    hipError_t cures = hipStreamSynchronize(starpu_cuda_get_local_stream());
	if (cures)
		STARPU_CUDA_REPORT_ERROR(cures);
}

static __global__ void cuda_redux(real_t *ePot, real_t *ePot_worker){
    *ePot = *ePot + *ePot_worker;
    return;
}

extern "C" void ePot_redux_cuda_func(void *descr[], void *cl_arg){
    real_t *ePot = (real_t *)STARPU_VARIABLE_GET_PTR(descr[0]);
	real_t *ePot_worker = (real_t *)STARPU_VARIABLE_GET_PTR(descr[1]);

	cuda_redux<<<1,1, 0, starpu_cuda_get_local_stream()>>>(ePot, ePot_worker);
    hipStreamSynchronize(starpu_cuda_get_local_stream());
}

extern "C" void ePot_init_cuda_func(void *descr[], void *cl_arg){
    real_t *ePot = (real_t *)STARPU_VARIABLE_GET_PTR(descr[0]);
	hipMemsetAsync(ePot, 0, sizeof(real_t), starpu_cuda_get_local_stream());
}
