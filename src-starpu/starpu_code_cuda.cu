#include "hip/hip_runtime.h"
#include <starpu.h>
#include "starpu_code.h"

static __global__ void do_ljForce(
    real_t s6, real_t eShift, real_t epsilon, real_t rCut2, int nNbrBoxes, int nLocalBoxes, int* boxes, int* nAtoms, real3* r, real3* f, real_t* U, real_t* ePot
){
    int slices = (nLocalBoxes + NWORKERS - 1) / NWORKERS;

    for (int iBox = id * slices; iBox < (id + 1) * slices && iBox < nLocalBoxes; iBox++){

      int nIBox = nAtoms[iBox];
      
      // loop over neighbors of iBox
      for (int jTmp=0; jTmp<nNbrBoxes; jTmp++){
            int jBox = boxes[(iBox - id * slices) * nNbrBoxes + jTmp];
            
            //assert(jBox>=0);
            STARPU_ASSERT(jBox>=0);
            
            int nJBox = nAtoms[jBox];
            
            // loop over atoms in iBox
            for (int iOff=MAXATOMS*iBox; iOff<(iBox*MAXATOMS+nIBox); iOff++){

               // loop over atoms in jBox
               for (int jOff=jBox*MAXATOMS; jOff<(jBox*MAXATOMS+nJBox); jOff++){
                  real3 dr;
                  real_t r2 = 0.0;
                  for (int m=0; m<3; m++){
                     dr[m] = r[iOff][m]-r[jOff][m];
                     r2+=dr[m]*dr[m];
                  }

                  if ( r2 <= rCut2 && r2 > 0.0){

                     // Important note:
                     // from this point on r actually refers to 1.0/r
                     r2 = 1.0/r2;
                     real_t r6 = s6 * (r2*r2*r2);
                     real_t eLocal = r6 * (r6 - 1.0) - eShift;
                     U[iOff] += 0.5*eLocal;
                     // zona critica
                     *ePot += 0.5*eLocal;

                     // different formulation to avoid sqrt computation
                     real_t fr = - 4.0*epsilon*r6*r2*(12.0*r6 - 6.0);
                     for (int m=0; m<3; m++)
                           f[iOff][m] -= dr[m]*fr;
                  }
               } // loop over atoms in jBox
         } // loop over atoms in iBox
      } // loop over neighbor boxes
    }
}

extern "C" void gpu_func(void *buffers[], void *cl_arg){
    // gathering paramethers
    struct params *params = cl_arg;
    real_t       s6 = params->s6;
    real_t   eShift = params->eShift;
    real_t  epsilon = params->epsilon;
    real_t    rCut2 = params->rCut2;
    int          id = params->id;
    int   nNbrBoxes = params->nNbrBoxes;
    int nLocalBoxes = params->nLocalBoxes;

    //printf("%d\n", starpu_worker_get_id());

    // gathering buffers
    int*   boxes = (   int*) STARPU_VECTOR_GET_PTR(buffers[0]);
    int*  nAtoms = (   int*) STARPU_VECTOR_GET_PTR(buffers[1]);
    real3*     r = ( real3*) STARPU_VECTOR_GET_PTR(buffers[2]);
    real3*     f = ( real3*) STARPU_VECTOR_GET_PTR(buffers[3]);
    real_t*    U = (real_t*) STARPU_VECTOR_GET_PTR(buffers[4]);
    real_t* ePot = (real_t*) STARPU_VARIABLE_GET_PTR(buffers[5]);
    
	do_ljForce<<<1, 1, 0, starpu_cuda_get_local_stream()>>>(s6, eShift, epsilon, rCut2, nNbrBoxes, nLocalBoxes, boxes, nAtoms, r, f, U, ePot);
    hipError_t cures = hipStreamSynchronize(starpu_cuda_get_local_stream());
	if (cures)
		STARPU_CUDA_REPORT_ERROR(cures);
}